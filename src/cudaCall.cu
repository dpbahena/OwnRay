#include "hip/hip_runtime.h"
#include "cudaCall.h"
#include "my_cudahelpers.h"
#include "my_classes.h"
#include "renderer.hpp"

#include <iostream>




namespace rayos {

    // #define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

    // void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    //     if (result) {
    //         std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
    //             file << ":" << line << " '" << func << "' \n";
    //         // Make sure we call CUDA Device Reset before exiting
    //         hipDeviceReset();
    //         exit(99);
    //     }
    // }

    #define checkCudaErrors(result) { gpuAssert((result), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
        if (code != hipSuccess) {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) assert(code == hipSuccess);
        }
    }


    __global__ void init_random(unsigned int seed, hiprandState_t* states){
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, idx, 0, &states[idx]);
    }

    __global__ void rand_init(hiprandState *rand_state) {
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            hiprand_init(1984, 0, 0, rand_state);
        }
    }


    __global__ void createWorld(/* sphere** d_sphere, */ hittable** list, hittable** world){
        if (threadIdx.x == 0 && blockIdx.x == 0){

            *(list)     = new sphere(vec3(0.0f, 0.0f, -1.0f), 0.5f);
            *(list+1)   = new sphere(vec3(0.0f, -100.5f, -1.0f), 100.0f);
            *world      = new hittable_list(list, 2);  // the list has 2 spheres
            
            
            
        }
    }

    __global__ void render_init(int max_x, int max_y, unsigned int seed, hiprandState *rand_state) {
        // int i = threadIdx.x + blockIdx.x * blockDim.x;
        // int j = threadIdx.y + blockIdx.y * blockDim.y;
        // if((i >= max_x) || (j >= max_y)) return;
        // int pixel_index = j*max_x + i;

        // Original: Each thread gets same seed, a different sequence number, no offset
        // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
        // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
        // performance improvement of about 2x!
        // hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, idx, 0, &rand_state[idx]);
    }

    __global__ void render_kernel(uint32_t* buffer, int width, int height, vec3 cameraCenter, vec3 delta_u, vec3 delta_v, vec3 pixel00, int samples, float scale, hittable** world, hiprandState_t* states){
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        int idx = width * j + i;
        // if (i == 0 && j == 0)
        //     printf("samples: %d\t", samples);
        if (idx >= (width * height)) return;
        vec3 color = vec3(0.0f, 0.0f, 0.0f);
        for (int x = 0; x < samples; x++){
            ray r = get_ray(i, j, pixel00, cameraCenter, delta_u, delta_v, states);
            color += ray_color(r, world);
        }

        // auto pixel_center = pixel00 + (static_cast<float>(i) * delta_u) + (static_cast<float>(j) * delta_v);
        // auto ray_direction = pixel_center - cameraCenter;
        // ray r(cameraCenter, ray_direction);

        // vec3 color = ray_color(r, world);
        color *= scale;
        buffer[idx] = colorToUint32_t(color);
        
    }

    __global__ void render_kernel2(uint32_t* buffer, int width, int height, vec3 cameraCenter, vec3 delta_u, vec3 delta_v, vec3 pixel00, int samples, float scale, hittable** world, hiprandState* rand_state){
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        int idx = width * j + i;
        hiprandState local_rand_state = rand_state[idx];
        if (i == 0 && j == 0)
            printf("samples: %d\t", samples);
        if (idx >= (width * height)) return;
        vec3 color = vec3(0.0f, 0.0f, 0.0f);
        for (int x = 0; x < samples; x++){
            float u = float(i + hiprand_uniform(&local_rand_state)) / float(width);
            float v = float(j + hiprand_uniform(&local_rand_state)) / float(height);
            ray r = get_ray2(i, j, pixel00, cameraCenter, delta_u, delta_v, u, v);
            color += ray_color(r, world);
        }

        // auto pixel_center = pixel00 + (static_cast<float>(i) * delta_u) + (static_cast<float>(j) * delta_v);
        // auto ray_direction = pixel_center - cameraCenter;
        // ray r(cameraCenter, ray_direction);

        // vec3 color = ray_color(r, world);
        color *= scale;
        buffer[idx] = colorToUint32_t(color);
        
    }

    __global__ void freeWorld(hittable** list, hittable** world){
        // delete buffer;
        delete *(list);
        delete *(list + 1);
        delete *world;
    }


    void CudaCall::cudaCall(int width, int height, Data& data)
    {
        
        Renderer renderer{window};
        uint32_t* colorBuffer;  
        checkCudaErrors(hipMallocManaged(&colorBuffer, width * height * sizeof(uint32_t)));

        
        // Create world memory

        hittable** d_list;
        checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hittable*)) );
        hittable** d_world;
        checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*) ));

       
        createWorld<<<1, 1>>>(d_list, d_world);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize() );

        



        clock_t start, stop;
        start = clock();
        int threads = 32;
        dim3 blockSize(threads, threads);
        int blocks_x = (width + blockSize.x - 1) / blockSize.x;
        int blocks_y = (height + blockSize.y - 1) / blockSize.y;
        dim3 gridSize(blocks_x, blocks_y);

        //generate random seed to be used in rayTracer kernel
        int num_threads = threads * threads * blocks_x * blocks_y;
        hiprandState_t* d_states;
        checkCudaErrors(hipMalloc((void**)&d_states, num_threads * sizeof(hiprandState_t)) );
        init_random<<<gridSize, blockSize>>>(time(0), d_states);

        // hiprandState* r_state;
        // checkCudaErrors(hipMalloc((void**)&r_state, num_threads * sizeof(hiprandState)) );
        // render_init<<<gridSize, blockSize>>>(width, height, time(0), r_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize() );


        render_kernel<<<gridSize, blockSize>>>(colorBuffer, width, height, data.center, data.delta_u, data.delta_v, data.pixel000, data.samples, data.scale, d_world, d_states);
        // render_kernel2<<<gridSize, blockSize>>>(colorBuffer, width, height, data.center, data.delta_u, data.delta_v, data.pixel000, data.samples, data.scale, d_world, r_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize() );
        stop = clock();
        double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
        std::cerr << "took " << timer_seconds << " seconds.\n";


        renderer.render(colorBuffer);


        freeWorld<<<1, 1>>>(d_list, d_world);
        hipFree(colorBuffer);
        
        hipFree(d_list);
        hipFree(d_world);
        hipFree(d_states);
        // hipFree(r_state);

    }

    CudaCall::~CudaCall()
    {
    }

     rayos::CudaCall::CudaCall(Window& window) : window(window)
    {
    }

    

} // namespace